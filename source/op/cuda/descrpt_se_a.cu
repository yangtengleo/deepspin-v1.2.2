#include "hip/hip_runtime.h"
/* Copyright 2015 The TensorFlow Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/
#define EIGEN_USE_GPU
#include <vector>
#include <climits>
#include <stdio.h>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <hip/hip_runtime.h>

#ifdef HIGH_PREC
    typedef double  VALUETYPE;
#else
    typedef float   VALUETYPE;
#endif

typedef unsigned long long int_64;

#define cudaErrcheck(res) { cudaAssert((res), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"cuda assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

template <
    typename    Key,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
__launch_bounds__ (BLOCK_THREADS)
__global__ void BlockSortKernel(
    Key * d_in,
    Key * d_out)            // Tile of output
{   
    enum { TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD };
    // Specialize BlockLoad type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
    typedef hipcub::BlockLoad<Key, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;
    // Specialize BlockRadixSort type for our thread block
    typedef hipcub::BlockRadixSort<Key, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
    // Shared memory
    __shared__ union TempStorage
    {
        typename BlockLoadT::TempStorage        load;
        typename BlockRadixSortT::TempStorage   sort;
    } temp_storage;
    // Per-thread tile items
    Key items[ITEMS_PER_THREAD];
    // Our current block's offset
    int block_offset = blockIdx.x * TILE_SIZE;
    // Load items into a blocked arrangement
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, items);
    // Barrier for smem reuse
    __syncthreads();
    // Sort keys
    BlockRadixSortT(temp_storage.sort).SortBlockedToStriped(items);
    // Store output in striped fashion
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_out + block_offset, items);
}

template<typename T>
__device__ inline T dev_dot(T * arr1, T * arr2) {
    return arr1[0] * arr2[0] + arr1[1] * arr2[1] + arr1[2] * arr2[2];
}

template<typename FPTYPE>
__device__ inline void spline5_switch(FPTYPE & vv,
        FPTYPE & dd,
        FPTYPE & xx, 
		const float & rmin, 
		const float & rmax) 
{
    if (xx < rmin) {
        dd = 0;
        vv = 1;
    }
    else if (xx < rmax) {
        FPTYPE uu = (xx - rmin) / (rmax - rmin) ;
        FPTYPE du = 1. / (rmax - rmin) ;
        vv = uu*uu*uu * (-6 * uu*uu + 15 * uu - 10) + 1;
        dd = ( 3 * uu*uu * (-6 * uu*uu + 15 * uu - 10) + uu*uu*uu * (-12 * uu + 15) ) * du;
    }
    else {
        dd = 0;
        vv = 0;
    }
}

__global__ void get_i_idx_se_a(const int nloc,
                        const int * ilist,
                        int * i_idx)
{
    const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;
    if(idy >= nloc) {
        return;
    }
    i_idx[ilist[idy]] = idy;
}

__global__ void format_nlist_fill_a_se_a(const VALUETYPE * coord,
                            const int * type,
                            const int  * jrange,
                            const int  * jlist,
                            const float rcut,
                            int_64 * key,
                            int * i_idx,
                            const int MAGIC_NUMBER)
{   
    // <<<nloc, MAGIC_NUMBER>>>
    const unsigned int idx = blockIdx.y;
    const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;
    
    const int nsize = jrange[i_idx[idx] + 1] - jrange[i_idx[idx]];
    if (idy >= nsize) {
        return;
    }

    const int * nei_idx = jlist + jrange[i_idx[idx]];
    // dev_copy(nei_idx, &jlist[jrange[i_idx]], nsize);

    int_64 * key_in = key + idx * MAGIC_NUMBER;

    VALUETYPE diff[3];
    const int & j_idx = nei_idx[idy];
    for (int dd = 0; dd < 3; dd++) {
        diff[dd] = coord[j_idx * 3 + dd] - coord[idx * 3 + dd];
    }
    VALUETYPE rr = sqrt(dev_dot(diff, diff)); 
    if (rr <= rcut) {
        key_in[idy] = type[j_idx] * 1E15+ (int_64)(rr * 1.0E13) / 100000 * 100000 + j_idx;
    }
}

    // bubble_sort(sel_nei, num_nei);
__global__ void format_nlist_fill_b_se_a(int * nlist,
                            const int nlist_size,
                            const int nloc,
                            const int * jrange,
                            const int * jlist,
                            int_64 * key,
                            const int * sec_a,
                            const int sec_a_size,
                            int * nei_iter_dev,
                            const int MAGIC_NUMBER)
{ 

    const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;

    if(idy >= nloc) {
        return;
    }
    
    int * row_nlist = nlist + idy * nlist_size;
    int * nei_iter = nei_iter_dev + idy * sec_a_size;
    int_64 * key_out = key + nloc * MAGIC_NUMBER + idy * MAGIC_NUMBER;

    for (int ii = 0; ii < sec_a_size; ii++) {
        nei_iter[ii] = sec_a[ii];
    }
    
    for (unsigned int kk = 0; key_out[kk] != key_out[MAGIC_NUMBER - 1]; kk++) {
        const int & nei_type = key_out[kk] / 1E15;
        if (nei_iter[nei_type] < sec_a[nei_type + 1]) {
            row_nlist[nei_iter[nei_type]++] = key_out[kk] % 100000;
        }
    }
}
//it's ok!

__global__ void compute_descriptor_se_a (VALUETYPE* descript,
                            const int ndescrpt,
                            VALUETYPE* descript_deriv,
                            const int descript_deriv_size,
                            VALUETYPE* rij,
                            const int rij_size,
                            const int* type,
                            const VALUETYPE* avg,
                            const VALUETYPE* std,
                            int* nlist,
                            const int nlist_size,
                            const VALUETYPE* coord,
                            const float rmin,
                            const float rmax,
                            const int sec_a_size)
{   
    // <<<nloc, sec_a.back()>>>
    const unsigned int idx = blockIdx.y;
    const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_deriv = idy * 4 * 3;	// 4 components time 3 directions
    const int idx_value = idy * 4;	// 4 components
    if (idy >= sec_a_size) {return;}

    // else {return;}
    VALUETYPE * row_descript = descript + idx * ndescrpt;
    VALUETYPE * row_descript_deriv = descript_deriv + idx * descript_deriv_size;
    VALUETYPE * row_rij = rij + idx * rij_size;
    int * row_nlist = nlist + idx * nlist_size;

    if (row_nlist[idy] >= 0) {
        const int & j_idx = row_nlist[idy];
        for (int kk = 0; kk < 3; kk++) {
            row_rij[idy * 3 + kk] = coord[j_idx * 3 + kk] - coord[idx * 3 + kk];
        }
        const VALUETYPE * rr = &row_rij[idy * 3 + 0];
        VALUETYPE nr2 = dev_dot(rr, rr);
        VALUETYPE inr = 1./sqrt(nr2);
        VALUETYPE nr = nr2 * inr;
        VALUETYPE inr2 = inr * inr;
        VALUETYPE inr4 = inr2 * inr2;
        VALUETYPE inr3 = inr4 * nr;
        VALUETYPE sw, dsw;
        spline5_switch(sw, dsw, nr, rmin, rmax);
        row_descript[idx_value + 0] = (1./nr)       ;//* sw;
        row_descript[idx_value + 1] = (rr[0] / nr2) ;//* sw;
        row_descript[idx_value + 2] = (rr[1] / nr2) ;//* sw;
        row_descript[idx_value + 3] = (rr[2] / nr2) ;//* sw;

        row_descript_deriv[idx_deriv + 0] = (rr[0] * inr3 * sw - row_descript[idx_value + 0] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 0) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 0) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 1] = (rr[1] * inr3 * sw - row_descript[idx_value + 0] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 1) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 1) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 2] = (rr[2] * inr3 * sw - row_descript[idx_value + 0] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 2) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 2) % (ndescrpt * 3)) / 3];
        // ****deriv of component x/r2
        row_descript_deriv[idx_deriv + 3] = ((2. * rr[0] * rr[0] * inr4 - inr2) * sw - row_descript[idx_value + 1] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 3) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 3) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 4] = ((2. * rr[0] * rr[1] * inr4	) * sw - row_descript[idx_value + 1] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 4) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 4) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 5] = ((2. * rr[0] * rr[2] * inr4	) * sw - row_descript[idx_value + 1] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 5) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 5) % (ndescrpt * 3)) / 3];
        // ***deriv of component y/r2
        row_descript_deriv[idx_deriv + 6] = ((2. * rr[1] * rr[0] * inr4	) * sw - row_descript[idx_value + 2] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 6) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 6) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 7] = ((2. * rr[1] * rr[1] * inr4 - inr2) * sw - row_descript[idx_value + 2] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 7) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 7) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 8] = ((2. * rr[1] * rr[2] * inr4	) * sw - row_descript[idx_value + 2] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 8) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 8) % (ndescrpt * 3)) / 3];
        // ***deriv of component z/r2
        row_descript_deriv[idx_deriv + 9] = ((2. * rr[2] * rr[0] * inr4	) * sw - row_descript[idx_value + 3] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 9) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 9) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv +10] = ((2. * rr[2] * rr[1] * inr4	) * sw - row_descript[idx_value + 3] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 10) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 10) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv +11] = ((2. * rr[2] * rr[2] * inr4 - inr2) * sw - row_descript[idx_value + 3] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 11) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 11) % (ndescrpt * 3)) / 3];
        // 4 value components
        row_descript[idx_value + 0] *= sw; // * descript[idx * ndescrpt + idx_value + 0]);// - avg[type[idx] * ndescrpt + idx_value + 0]) / std[type[idx] * ndescrpt + idx_value + 0];
        row_descript[idx_value + 1] *= sw; // * descript[idx * ndescrpt + idx_value + 1]);// - avg[type[idx] * ndescrpt + idx_value + 1]) / std[type[idx] * ndescrpt + idx_value + 1];
        row_descript[idx_value + 2] *= sw; // * descript[idx * ndescrpt + idx_value + 2]);// - avg[type[idx] * ndescrpt + idx_value + 2]) / std[type[idx] * ndescrpt + idx_value + 2];
        row_descript[idx_value + 3] *= sw; // * descript[idx * ndescrpt + idx_value + 3]);// - avg[type[idx] * ndescrpt + idx_value + 3]) / std[type[idx] * ndescrpt + idx_value + 3];
    }

    for (int ii = 0; ii < 4; ii++) {
        row_descript[idx_value + ii] = (row_descript[idx_value + ii] - avg[type[idx] * ndescrpt + idx_value + ii]) / std[type[idx] * ndescrpt + idx_value + ii];
    }
    // idy nloc, idx ndescrpt * 3
    // descript_deriv[idy * ndescrpt * 3 + idx] = (descript_deriv_dev[idy * (ndescrpt * 3) + idx]) / std[type[idy] * ndescrpt + idx / 3];
    for (int ii = 0; ii < 12; ii++) {
        row_descript_deriv[idx_deriv + ii] /= std[type[idx] * ndescrpt + (idx_deriv + ii) / 3];
    }
}

void format_nbor_list_256 (
    const VALUETYPE* coord,
    const int* type,
    const int* jrange,
    const int* jlist,
    const int& nloc,       
    const float& rcut_r, 
    int * i_idx, 
    int_64 * key
) 
{   
    const int LEN = 256;
    const int MAGIC_NUMBER = 256;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nblock, nloc);
    format_nlist_fill_a_se_a
    <<<block_grid, LEN>>> (
        coord,
        type,
        jrange,
        jlist,
        rcut_r,
        key,
        i_idx,
        MAGIC_NUMBER
    );
    const int ITEMS_PER_THREAD = 4;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    // BlockSortKernel<NeighborInfo, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>> (
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

void format_nbor_list_512 (
    const VALUETYPE* coord,
    const int* type,
    const int* jrange,
    const int* jlist,
    const int& nloc,       
    const float& rcut_r, 
    int * i_idx, 
    int_64 * key
) 
{   
    const int LEN = 256;
    const int MAGIC_NUMBER = 512;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nblock, nloc);
    format_nlist_fill_a_se_a
    <<<block_grid, LEN>>> (
        coord,
        type,
        jrange,
        jlist,
        rcut_r,
        key,
        i_idx,
        MAGIC_NUMBER
    );
    const int ITEMS_PER_THREAD = 4;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    // BlockSortKernel<NeighborInfo, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>> (
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

void format_nbor_list_1024 (
    const VALUETYPE* coord,
    const int* type,
    const int* jrange,
    const int* jlist,
    const int& nloc,       
    const float& rcut_r, 
    int * i_idx, 
    int_64 * key
) 
{   
    const int LEN = 256;
    const int MAGIC_NUMBER = 1024;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nblock, nloc);
    format_nlist_fill_a_se_a
    <<<block_grid, LEN>>> (
        coord,
        type,
        jrange,
        jlist,
        rcut_r,
        key,
        i_idx,
        MAGIC_NUMBER
    );
    const int ITEMS_PER_THREAD = 8;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    // BlockSortKernel<NeighborInfo, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>> (
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

void format_nbor_list_2048 (
    const VALUETYPE* coord,
    const int* type,
    const int* jrange,
    const int* jlist,
    const int& nloc,       
    const float& rcut_r, 
    int * i_idx, 
    int_64 * key
) 
{   
    const int LEN = 256;
    const int MAGIC_NUMBER = 2048;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nblock, nloc);
    format_nlist_fill_a_se_a
    <<<block_grid, LEN>>> (
        coord,
        type,
        jrange,
        jlist,
        rcut_r,
        key,
        i_idx,
        MAGIC_NUMBER
    );
    const int ITEMS_PER_THREAD = 8;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    // BlockSortKernel<NeighborInfo, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>> (
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

void format_nbor_list_4096 (
    const VALUETYPE* coord,
    const int* type,
    const int* jrange,
    const int* jlist,
    const int& nloc,       
    const float& rcut_r, 
    int * i_idx, 
    int_64 * key
) 
{   
    const int LEN = 256;
    const int MAGIC_NUMBER = 4096;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nblock, nloc);
    format_nlist_fill_a_se_a
    <<<block_grid, LEN>>> (
        coord,
        type,
        jrange,
        jlist,
        rcut_r,
        key,
        i_idx,
        MAGIC_NUMBER
    );
    const int ITEMS_PER_THREAD = 16;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    // BlockSortKernel<NeighborInfo, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>> (
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

void DescrptSeALauncher(const VALUETYPE* coord,
                            const int* type,
                            const int* ilist,
                            const int* jrange,
                            const int* jlist,
                            int* array_int,
                            unsigned long long* array_longlong,
                            const VALUETYPE* avg,
                            const VALUETYPE* std,
                            VALUETYPE* descript,
                            VALUETYPE* descript_deriv,
                            VALUETYPE* rij,
                            int* nlist,
                            const int& nloc,       
                            const int& nnei,       
                            const float& rcut_r,     
                            const float& rcut_r_smth,
                            const int& ndescrpt, 
                            const std::vector<int>& sec_a,      
                            const bool& fill_nei_a,
                            const int MAGIC_NUMBER
)
{   
    const int LEN = 256;
    int nblock = (nloc + LEN -1) / LEN;
    int * sec_a_dev = array_int;
    int * nei_iter = array_int + sec_a.size(); // = new int[sec_a_size];
    int * i_idx = array_int + sec_a.size() + nloc * sec_a.size();
    int_64 * key = array_longlong;
    
    hipError_t res = hipSuccess;
    res = hipMemcpy(sec_a_dev, &sec_a[0], sizeof(int) * sec_a.size(), hipMemcpyHostToDevice); cudaErrcheck(res);    
    res = hipMemset(key, 0xffffffff, sizeof(int_64) * nloc * MAGIC_NUMBER); cudaErrcheck(res);
    res = hipMemset(nlist, -1, sizeof(int) * nloc * nnei); cudaErrcheck(res);
    res = hipMemset(descript, 0.0, sizeof(VALUETYPE) * nloc * ndescrpt); cudaErrcheck(res);
    res = hipMemset(descript_deriv, 0.0, sizeof(VALUETYPE) * nloc * ndescrpt * 3); cudaErrcheck(res);

    if (fill_nei_a) {
        // ~~~
        // hipProfilerStart();
        get_i_idx_se_a<<<nblock, LEN>>> (nloc, ilist, i_idx);

        if (nnei <= 256) {
            format_nbor_list_256 (
                coord,
                type,
                jrange,
                jlist,
                nloc,       
                rcut_r, 
                i_idx, 
                key
            ); 
        } else if (nnei <= 512) {
            format_nbor_list_512 (
                coord,
                type,
                jrange,
                jlist,
                nloc,       
                rcut_r, 
                i_idx, 
                key
            ); 
        } else if (nnei <= 1024) {
            format_nbor_list_1024 (
                coord,
                type,
                jrange,
                jlist,
                nloc,       
                rcut_r, 
                i_idx, 
                key
            ); 
        } else if (nnei <= 2048) {
            format_nbor_list_2048 (
                coord,
                type,
                jrange,
                jlist,
                nloc,       
                rcut_r, 
                i_idx, 
                key
            ); 
        } else if (nnei <= 4096) {
            format_nbor_list_4096 (
                coord,
                type,
                jrange,
                jlist,
                nloc,       
                rcut_r, 
                i_idx, 
                key
            ); 
        } 

        format_nlist_fill_b_se_a<<<nblock, LEN>>> (
                            nlist,
                            nnei,       
                            nloc,
                            jrange,
                            jlist,
                            key,
                            sec_a_dev,
                            sec_a.size(),
                            nei_iter,
                            MAGIC_NUMBER
        );
    }

    const int nblock_ = (sec_a.back() + LEN -1) / LEN;
    dim3 block_grid(nblock_, nloc);
    compute_descriptor_se_a<<<block_grid, LEN>>> (
                            descript,
                            ndescrpt,
                            descript_deriv,
                            ndescrpt * 3,
                            rij,
                            nnei * 3,
                            type,
                            avg,
                            std,
                            nlist,
                            nnei,
                            coord,
                            rcut_r_smth,
                            rcut_r,
                            sec_a.back()
    );
}  